#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "include/graph.cuh"
#include <math.h>

#define CUDA_ERR_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

using namespace std;

__device__ void find_shortest_paths( 
    int *R, int *C, int *d, int *sigma, int *Q_curr, int *Q_next, int *S, int *S_ends, 
    int &Q_curr_len, int &Q_next_len, int &S_len, int &S_ends_len, int &depth, int &id, int &bsize) {

    // int id = threadIdx.x;
    // int bsize = blockDim.x;
    int i, j, v, w, last;

    while(true) {
        for(i = id; i < Q_curr_len; i += bsize) {
            v = Q_curr[i];
            for(j = R[v]; j < R[v+1]; j++) {
                w = C[j];
                if(atomicCAS(&d[w], -1, d[v] + 1) < 0) {
                    last = atomicAdd(&Q_next_len, 1);
                    Q_next[last] = w;
                }
                if(d[w] == (d[v] + 1)) {
                    atomicAdd(&sigma[w], sigma[v]);
                }
            }
        }

        __syncthreads();

        if(Q_next_len == 0) {
            if(id == 0)
                depth = d[S[S_len-1]] - 1;
            break;
        }
        else {
            for(i = id; i < Q_next_len; i += bsize) {
                Q_curr[i] = Q_next[i];
                S[i + S_len] = Q_next[i];
            }

            __syncthreads();

            if(id == 0) {
                S_ends[S_ends_len] = S_ends[S_ends_len-1] + Q_next_len;
                S_ends_len = S_ends_len + 1;
                Q_curr_len = Q_next_len;
                S_len = S_len + Q_next_len;
                Q_next_len = 0;
            }

            __syncthreads();
        }
    }
}

__device__ void accumulate_dependencies( int *R, int *C, int *d, int *sigma, float *delta, int *S, int *S_ends, int &depth, int &id, int &bsize) {

    // int id = threadIdx.x;
    // int bsize = blockDim.x;
    int i, j, v, w;
    float sw, sv, dsw;

    while(depth > 0) {
        for (i = id + S_ends[depth]; i < S_ends[depth+1]; i += bsize) {
            w = S[i];
            dsw = 0;
            sw = sigma[w];

            for(j = R[w]; j < R[w+1]; j++) {
                v = C[j];
                sv = sigma[v];
                if(d[v] = d[w] + 1) {
                    dsw += ((sw/sv)*(1+delta[v]));
                }
            }

            delta[w] = dsw;
        }

        __syncthreads();
        if(id == 0) depth--;
        __syncthreads();
    }
}

__global__ void brandes_parallel(int *R, int *C, int *d, int *sigma, float *delta, int s, int n_nodes, int max_nodes_in_level) {

    int id = threadIdx.x;
    __shared__ int bsize = blockDim.x;
    
    //(log2(4*n_nodes + 1)/log2(5)) + 1
    __shared__ int Q_curr[n_nodes];
    __shared__ int Q_curr_len;
    __shared__ int Q_next[n_nodes];
    __shared__ int Q_next_len;
    __shared__ int S[n_nodes];
    __shared__ int S_len;
    __shared__ int S_ends[n_nodes];
    __shared__ int S_ends_len;

    int v;
    for(v = id; id < n_nodes; v += bsize) {
        if(v == s) {
            d[v] = 0;
            sigma[v] = 1;
        }
        else {
            d[v] = -1;
            sigma[v] = 0;
        }
        delta[v] = 0;
    }

    if(id == 0) {
        Q_curr[0] = s;
        Q_curr_len = 1;
        Q_next_len = 0;
        S[0] = s;
        S_len = 1;
        S_ends[0] = 0;
        S_ends[1] = 1;
        S_ends_len = 2;
        depth = 0;
    }

    __syncthreads();
    
    find_shortest_paths( R, C, d, sigma, Q_curr, Q_next, S, S_ends, Q_curr_len, Q_next_len, S_len, S_ends_len, depth, id, bsize);
    __syncthreads();

    accumulate_dependencies(R, C, d, sigma, delta, S, S_ends, depth, id, bsize);
    __syncthreads();
}